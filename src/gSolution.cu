#include "hip/hip_runtime.h"
#include "gSolution.cuh"

const int nBlocks =2;
const int nThreads = 100;
const int maxChain = 10;
//remove sizeTabu of parameters in gSolution.cuh and TS-GAP.cu

__global__ void TS_GAP(Instance *inst, Solution *sol, EjectionChain *ejection, int *tabuListshort, unsigned int *seed, hiprandState_t* states, int iteration, int n_busca){
	//variables of auxiliars
	int i, j,k,flag, aux, aux_2;
	//use for counting amount resources
	int res_aux[100];
	//Solution of block in memory shared
	__shared__ short int s_shared[1600];
//	s_shared = (short int *)malloc(sizeof(short int)*inst->nJobs);
	//Iterator of size search
	int s_search;
	int term = threadIdx.x + blockIdx.x*nThreads;
	int tPos = threadIdx.x*maxChain + blockIdx.x*maxChain*nThreads;

	//save best Delta of ejection chain
	int d_best_aux;
	//save best Delta of iteration
	int delta_best;
	//save best ejection chain
	short int pos_best[maxChain];
	short int size_best;
	short int op_best;
	//Delta for each step of the ejection chain 
	int delta_aux[maxChain];
	//save best size of ejection chain
	int size_aux;


	hiprand_init(seed[blockIdx.x*nThreads + threadIdx.x],blockIdx.x*nThreads + threadIdx.x,0,&states[blockIdx.x*nThreads + threadIdx.x]);
	aux = 0;
	aux_2 = inst->nJobs/nThreads;
	for(i = 0; i<= aux_2;i++){
		aux = inst->nJobs - i*nThreads; 
		if(threadIdx.x < aux){
			s_shared[threadIdx.x + i*nThreads] = ((int)sol->s[threadIdx.x + i*nThreads + blockIdx.x*inst->nJobs]);
			__syncthreads();
		}
		__syncthreads();
	}
	__syncthreads();
	
//	if((threadIdx.x == 0)&&(blockIdx.x==0)){
//		for(i=0;i<inst->nJobs;i++){
//			printf("job %d  agent %d\n", i,s_shared[i]);
//		}
		
//	}
	
	for(s_search = 0; s_search < n_busca; s_search++){
		do{
			aux = 0;
			for(i=0; i< inst->mAgents; i++){
				res_aux[i] = sol->resUsage[i + blockIdx.x*inst->mAgents]; 
			}
			ejection->delta[term] = 0;
			ejection->op[term] = hiprand(&states[term])%2;
			if(ejection->op[term] == 1){

				ejection->sizeChain[term] = 0;
				ejection->pos[0 + tPos] = hiprand(&states[term])%inst->nJobs;
				ejection->pos[1 + tPos] = hiprand(&states[term])%inst->mAgents;
				if((sol->resUsage[ejection->pos[1 + tPos] + blockIdx.x*inst->mAgents] + inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + ejection->pos[ 1 + tPos]] <= inst->capacity[ejection->pos[1 + tPos]])
						&&(tabuListshort[ejection->pos[0+tPos] + blockIdx.x*inst->nJobs]<=iteration))
				{
					ejection->delta[term] = inst->cost[ejection->pos[0 + tPos]*inst->mAgents + ejection->pos[1 + tPos]] - inst->cost[ejection->pos[0 + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])];
					res_aux[ejection->pos[1 + tPos] + blockIdx.x*inst->mAgents] += inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + ejection->pos[ 1 + tPos]];
					res_aux[s_shared[ejection->pos[0 + tPos]]] -= inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + s_shared[ejection->pos[0 + tPos]]];
					aux = 1;
				}
				d_best_aux = ejection->delta[term];
			}else{
				aux = 1;
				ejection->sizeChain[term] = maxChain;
				size_aux = 2;
				d_best_aux = 1000000;

				//choosing first job what not tabu list
				do{
					ejection->pos[0 + tPos] = hiprand(&states[term])%inst->nJobs;
				}while(tabuListshort[ejection->pos[0+tPos] + blockIdx.x*inst->nJobs]>iteration);

				ejection->delta[term] -= inst->cost[ejection->pos[0 + tPos]*inst->mAgents + s_shared[ejection->pos[0+tPos]]];
				ejection->pos[(ejection->sizeChain[term]-1) + tPos] = inst->nJobs-1;
				for(i=1; i<ejection->sizeChain[term]; i++){
					do{
						ejection->pos[i + tPos] = hiprand(&states[term])%inst->nJobs;
					}while(tabuListshort[ejection->pos[i+tPos] + blockIdx.x*inst->nJobs]>iteration);
					k = ejection -> pos[i + tPos];
					do{
						flag = 0;
						aux_2 = 0;
						//a = 0; //descobrir para que serve;
						for(j=0;  j < i ; j++){
							if(ejection->pos[i + tPos]==ejection->pos[j + tPos]){
								flag = 1;
								break;
							}
						}
						__syncthreads();
						if((flag!=1)&&(s_shared[ ejection->pos[i + tPos] ] != s_shared[ ejection->pos[(i-1) + tPos] ])){
							if( sol->resUsage[s_shared[ejection->pos[i + tPos]]+blockIdx.x*inst->mAgents] - inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents+s_shared[ejection->pos[i + tPos]]] + inst->resourcesAgent[ejection->pos[(i-1)+tPos]*inst->mAgents+s_shared[ejection->pos[i + tPos]]] <= inst->capacity[s_shared[ejection->pos[i + tPos] ]]){
								__syncthreads();
								if( i == ejection->sizeChain[term]-1){
									if((s_shared[ejection->pos[0 + tPos]])!=(s_shared[ejection->pos[(ejection->sizeChain[term]-1) + tPos]])){
										//res_aux[s_shared[ejection->pos[i +tPos]]] -= inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + s_shared[ejection->pos[i +tPos]]];
										res_aux[s_shared[ejection->pos[i +tPos]]] += inst->resourcesAgent[ejection->pos[(i-1) +tPos]*inst->mAgents + s_shared[ejection->pos[i +tPos]]] - inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + s_shared[ejection->pos[i +tPos]]];
										ejection->delta[term] += inst->cost[ejection->pos[(i-1) + tPos]*inst->mAgents+s_shared[ejection->pos[i +tPos]]] - inst->cost[ejection->pos[i + tPos]*inst->mAgents+s_shared[ejection->pos[i +tPos]]]; //update delta
										//ejection->delta[term] -= inst->cost[ejection->pos[i + tPos]*inst->mAgents+s_shared[ejection->pos[i +tPos]]];
										if(sol->resUsage[s_shared[ejection->pos[0 + tPos]] + blockIdx.x*inst->mAgents] - inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + s_shared[ejection->pos[0 + tPos]]] + inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + s_shared[ejection->pos[0 + tPos]]] <= inst->capacity[s_shared[ejection->pos[0 + tPos]]]){
											delta_aux[i] = ejection->delta[term] + inst->cost[ejection->pos[i+tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])]; 
										}else{
											delta_aux[i] = 1000000;
										}
										if(delta_aux[i]<d_best_aux){
											d_best_aux = delta_aux[i];
											size_aux = i + 1;
										}
										aux_2 = 2;
										break;
									}
								}else{
									//res_aux[s_shared[ejection->pos[i +tPos]]] -= inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + s_shared[ejection->pos[i +tPos]]];
									res_aux[s_shared[ejection->pos[i +tPos]]] += inst->resourcesAgent[ejection->pos[(i-1) +tPos]*inst->mAgents + s_shared[ejection->pos[i +tPos]]] - inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + s_shared[ejection->pos[i +tPos]]];
									ejection->delta[term] += inst->cost[ejection->pos[(i-1) + tPos]*inst->mAgents+s_shared[ejection->pos[i +tPos]]] - inst->cost[ejection->pos[i + tPos]*inst->mAgents+s_shared[ejection->pos[i +tPos]]]; //update delta
									//ejection->delta[term] -= inst->cost[ejection->pos[i + tPos]*inst->mAgents+s_shared[ejection->pos[i +tPos]]];
									if(sol->resUsage[s_shared[ejection->pos[0 + tPos]] + blockIdx.x*inst->mAgents] - inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + s_shared[ejection->pos[0 + tPos]]] + inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + s_shared[ejection->pos[0 + tPos]]] <= inst->capacity[s_shared[ejection->pos[0 + tPos]]]){
										delta_aux[i] = ejection->delta[term] + inst->cost[ejection->pos[i+tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])]; 
									}else{
										delta_aux[i] = 1000000;
									}
									if(delta_aux[i]<d_best_aux){
										d_best_aux = delta_aux[i];
										size_aux = i + 1;
									}
									aux_2 = 2;
									break;
								}
							}
						}	
						do{		
							ejection->pos[i + tPos] = (ejection->pos[i + tPos]+1)%(inst->nJobs);
						}while((tabuListshort[ejection->pos[i+tPos] + blockIdx.x*inst->nJobs]>iteration)&&(ejection->pos[i + tPos]!=k));
					}while(ejection->pos[i + tPos]!=k);
					if(aux_2==0)
                                        {
                                                if((i>1)&&(s_shared[ejection->pos[0+tPos]]) != (s_shared[ejection->pos[(i-1)+tPos]]))             
                                                {
                                                        ejection->sizeChain[term] = i;
                                                }
                                                else
                                                {
                                                        aux = 0;
                                                }
                                                break;
                                        }
				}
  				ejection->sizeChain[term] = size_aux;
                                ejection->delta[term] = d_best_aux;
                                //ejection->delta[term] += inst->cost[ejection->pos[(ejection->sizeChain[term]-1)+tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])];//update with last and first position
//                                res_aux[s_shared[ejection->pos[0 +tPos]]] -= inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + s_shared[ejection->pos[0 +tPos]]];
                                res_aux[s_shared[ejection->pos[0 +tPos]]] += inst->resourcesAgent[ejection->pos[(ejection->sizeChain[term]-1) +tPos]*inst->mAgents + s_shared[ejection->pos[0 +tPos]]] -  inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + s_shared[ejection->pos[0 +tPos]]];

                                if((sol->resUsage[s_shared[ejection->pos[0 + tPos]]] - inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + s_shared[ejection->pos[0 + tPos]]] + inst->resourcesAgent[ejection->pos[(ejection->sizeChain[term]-1) +tPos]*inst->mAgents + s_shared[ejection->pos[0 + tPos]]]>inst->capacity[s_shared[ejection->pos[0 + tPos]]])||(s_shared[ejection->pos[0 + tPos]] == s_shared[ejection->pos[(ejection->sizeChain[term]-1) + tPos]]))
                                {
                                        aux=0;
                                }
                                for(i=0; i<inst->mAgents; i++)
                                {
                                        if(res_aux[i]>inst->capacity[i])
                                        {
                                                aux=0;
                                                break;
                                        }
                                }

			}

		}while((aux==0)||(d_best_aux== 1000000));
		if(s_search==0){
			op_best = ejection->op[term];
			delta_best = ejection->delta[term];
			for (i=0;i<maxChain;i++){
				pos_best[i] =  ejection->pos[i + tPos];
			}
			size_best = ejection->sizeChain[term];

		}else{
			if(ejection->delta[term]<delta_best){
				op_best = ejection->op[term];
				delta_best = ejection->delta[term];
				for (i=0;i<maxChain;i++){
					pos_best[i] =  ejection->pos[i + tPos];
				}
				size_best = ejection->sizeChain[term];
			}
		}

	}
	ejection->op[term] = op_best;
	ejection->delta[term] = delta_best;
	for (i=0;i<maxChain;i++){
		ejection->pos[i + tPos] = pos_best[i];
	}
	ejection->sizeChain[term] = size_best;
	__syncthreads();
//	if(threadIdx.x==0)
//		free(s_shared);
}





Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents)
{
	size_t size_solution = sizeof(Solution)
                        		   + sizeof(TcostFinal)*nBlocks
                        		   + sizeof(Ts)*(nJobs*nBlocks) //vector s
                        		   + sizeof(TresUsage)*(mAgents*nBlocks); // vector resUsage



	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	gpuMemset(d_sol,0,size_solution);
	h_solution->costFinal = (TcostFinal*)(d_sol+1);
	h_solution->s = (Ts*)(h_solution->costFinal + nBlocks);
	h_solution->resUsage = (TresUsage*)(h_solution->s + (nJobs*nBlocks));
	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);
	return d_sol;
}

EjectionChain* createGPUejection(EjectionChain* h_ejection,TnJobs nJobs, TmAgents mAgents)
{
	size_t size_ejection = sizeof(EjectionChain)
                        		   + sizeof(Tpos)*(nBlocks*nThreads*maxChain)
                        		   + sizeof(Top)*(nBlocks*nThreads)
                        		   + sizeof(TSizeChain)*(nBlocks*nThreads)
                        		   + sizeof(Tdelta)*(nBlocks*nThreads);
	EjectionChain *d_ejection;
	gpuMalloc((void**)&d_ejection, size_ejection);
	gpuMemset(d_ejection,0,size_ejection);
	h_ejection->pos=(Tpos*)(d_ejection + 1);
	h_ejection->op = (Top*)(h_ejection->pos+ (nBlocks*nThreads*maxChain));
	h_ejection->sizeChain = (TSizeChain*)(h_ejection->op + (nBlocks*nThreads));
	h_ejection->delta = (Tdelta*)(h_ejection->sizeChain + (nBlocks*nThreads));
	gpuMemcpy(d_ejection, h_ejection, size_ejection, hipMemcpyHostToDevice);
	return d_ejection;
}

