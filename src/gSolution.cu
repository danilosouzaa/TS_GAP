#include "hip/hip_runtime.h"
#include "gSolution.cuh"

const int nBlocks = 24;
const int nThreads = 576;
const int maxChain = 8;


__global__ void TS_GAP(Instance *inst, Solution *sol,EjectionChain *ejection, int *tabuListshort, unsigned int *seed, hiprandState_t* states, int iteration,int sizeTabu, int n_busca)
{
	int i,j,k,flag,a,w;
	//variable for verify if solution is feasible
	int aux = 0;
	int res_aux[80];
	int delta_aux[maxChain];
	int size_aux;
	int d_best_aux;
	int delta_best;
	short int pos_best[maxChain];
	short int size_best;
	short int op_best;

	//initializes of hiprand
	hiprand_init(seed[blockIdx.x*nThreads + threadIdx.x],blockIdx.x*nThreads + threadIdx.x,0,&states[blockIdx.x*nThreads + threadIdx.x]);
	int term = threadIdx.x + blockIdx.x*nThreads;
	int tPos = threadIdx.x*maxChain + blockIdx.x*maxChain*nThreads;
	for(w = 0; w<n_busca; w++)
	{
		do
		{
			for(i=0; i<inst->mAgents; i++)
			{
				res_aux[i]=0;
			}

			ejection->delta[term] = 0;
			ejection->op[term] = hiprand(&states[term])%2;
			__syncthreads();
			if(ejection->op[term] == 1)
			{
				ejection->sizeChain[term]=0;
				ejection->pos[0 + tPos] = hiprand(&states[term])%inst->nJobs;
				ejection->pos[1 + tPos] = hiprand(&states[term])%inst->mAgents;
				if((sol->resUsage[ejection->pos[1 + tPos] + blockIdx.x*inst->mAgents] + inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + ejection->pos[ 1 + tPos]] <= inst->capacity[ejection->pos[1 + tPos]])
						&&(tabuListshort[ejection->pos[0+tPos] + ejection->pos[1+tPos]*inst->nJobs + blockIdx.x*inst->nJobs*inst->mAgents]<iteration))
				{
					ejection->delta[term] = inst->cost[ejection->pos[0 + tPos]*inst->mAgents + ejection->pos[1 + tPos]] - inst->cost[ejection->pos[0 + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])];
					aux = 1;
				}
				//printf("Shift, with delta equals the %d\n",ejection->delta[term]);
			}
			else
			{

				aux = 1;
				ejection->sizeChain[term] = hiprand(&states[term])%(maxChain-1) + 2;
				size_aux = 2;
				d_best_aux = 1000000;
				ejection->pos[0 + tPos] = hiprand(&states[term])%inst->nJobs;
				ejection->delta[term] -= inst->cost[ejection->pos[0 + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0+tPos] + blockIdx.x*inst->nJobs])];
				ejection->pos[(ejection->sizeChain[term]-1) + tPos] = inst->nJobs-1;
				for(i=1; i<ejection->sizeChain[term]; i++)
				{
					ejection->pos[i + tPos] = hiprand(&states[term])%inst->nJobs;
					k = ejection -> pos[i + tPos];
					do
					{
						flag = 0;
						a = 0;

						for(j=0; j<i; j++)
						{
							if(ejection->pos[i + tPos]==ejection->pos[j + tPos])
							{
								flag = 1;
								break;
							}
						}

						if(((int)sol->s[ejection->pos[i + tPos] + blockIdx.x*inst->nJobs]) != ((int)sol->s[ejection->pos[(i-1) + tPos] + blockIdx.x*inst->nJobs]))
						{
							//						if(((int)sol->s[ejection->pos[i + tPos] + blockIdx.x*inst->nJobs])>4)
								//							printf("Agente: %d, Job: %d\n", ((int)sol->s[ejection->pos[i + tPos] + blockIdx.x*inst->nJobs]),ejection->pos[i + tPos]);
							if((flag!=1)&&(sol->resUsage[((int)sol->s[ejection->pos[i + tPos] + blockIdx.x*inst->nJobs]) + blockIdx.x*inst->mAgents] - inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[i + tPos] + blockIdx.x*inst->nJobs])] + inst->resourcesAgent[ejection->pos[(i-1) + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[i + tPos] + blockIdx.x*inst->nJobs])] <= inst->capacity[((int)sol->s[ejection->pos[i + tPos] + blockIdx.x*inst->nJobs])]))
							{
								if( tabuListshort[ ejection->pos[i-1 + tPos] + ((int)sol->s[ejection->pos[i + tPos] + blockIdx.x*inst->nJobs])*inst->nJobs + blockIdx.x*inst->nJobs*inst->mAgents]<iteration)
								{
									__syncthreads();
									if( i == ejection->sizeChain[term]-1)
									{
										if(((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs]) != ((int)sol->s[ejection->pos[(ejection->sizeChain[term]-1) + tPos] + blockIdx.x*inst->nJobs]))
										{
											if(tabuListshort[ ejection->pos[(ejection->sizeChain[term]-1) + tPos] + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])*inst->nJobs + blockIdx.x*inst->nJobs*inst->mAgents]<iteration )
											{
												res_aux[((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])] -= inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])];
												res_aux[((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])] += inst->resourcesAgent[ejection->pos[(i-1) +tPos]*inst->mAgents + ((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])];
												ejection->delta[term] += inst->cost[ejection->pos[(i-1) + tPos]*inst->mAgents+((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])];//update delta
												ejection->delta[term] -= inst->cost[ejection->pos[i + tPos]*inst->mAgents+((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])];

												if((sol->resUsage[((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs]) + blockIdx.x*inst->mAgents] - inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos]+blockIdx.x*inst->nJobs])] + inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])]<=inst->capacity[((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])])||(((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs]) == ((int)sol->s[ejection->pos[i + tPos]+blockIdx.x*inst->nJobs]))){
													delta_aux[i] = ejection->delta[term] + inst->cost[ejection->pos[i+tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])]; 
												}else{
													delta_aux[i] = 1000000;
												}
												if(delta_aux[i]<d_best_aux){
													d_best_aux = delta_aux[i];
													size_aux = i + 1;
												}
												a = 2;
												break; //if yes, next position is randomly selected
											}
										}
									}
									else
									{
										res_aux[((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])] -= inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])];
										res_aux[((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])] += inst->resourcesAgent[ejection->pos[(i-1) +tPos]*inst->mAgents + ((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])];
										ejection->delta[term] += inst->cost[ejection->pos[(i-1) + tPos]*inst->mAgents+((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])];//update delta
										ejection->delta[term] -= inst->cost[ejection->pos[i + tPos]*inst->mAgents+((int)sol->s[ejection->pos[i +tPos] + blockIdx.x*inst->nJobs])];
										if((sol->resUsage[((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs]) + blockIdx.x*inst->mAgents] - inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos]+blockIdx.x*inst->nJobs])] + inst->resourcesAgent[ejection->pos[i + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])]<=inst->capacity[((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])])||(((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs]) == ((int)sol->s[ejection->pos[i + tPos]+blockIdx.x*inst->nJobs]))){
											delta_aux[i] = ejection->delta[term] + inst->cost[ejection->pos[i+tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])];
										}else{
											delta_aux[i] = 1000000;
										}
										if(delta_aux[i]<d_best_aux){
											d_best_aux = delta_aux[i];
											size_aux = i + 1;
										}
										a = 2;
										break;
									}
								}
							}
						}
						ejection->pos[i + tPos] = (ejection->pos[i + tPos]+1)%(inst->nJobs);
					}
					while(ejection->pos[i + tPos]!=k);
					if(a==0)
					{
						if((i>1)&&(((int)sol->s[ejection->pos[0+tPos] + blockIdx.x*inst->nJobs]) != ((int)sol->s[ejection->pos[(i-1)+tPos]+blockIdx.x*inst->nJobs]))&&
								(tabuListshort[ ejection->pos[(i-1) + tPos] + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])*inst->nJobs + blockIdx.x*inst->nJobs*inst->mAgents]<iteration ))
						{
							ejection->sizeChain[term] = i;
						}
						else
						{
							aux = 0;

						}
						break;
					}
				}
				ejection->sizeChain[term] = size_aux;
				ejection->delta[term] = d_best_aux;
				//ejection->delta[term] += inst->cost[ejection->pos[(ejection->sizeChain[term]-1)+tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])];//update with last and first position
				res_aux[((int)sol->s[ejection->pos[0 +tPos] + blockIdx.x*inst->nJobs])] -= inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 +tPos] + blockIdx.x*inst->nJobs])];
				res_aux[((int)sol->s[ejection->pos[0 +tPos] + blockIdx.x*inst->nJobs])] += inst->resourcesAgent[ejection->pos[(ejection->sizeChain[term]-1) +tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 +tPos] + blockIdx.x*inst->nJobs])];
				if((sol->resUsage[((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs]) + blockIdx.x*inst->mAgents] - inst->resourcesAgent[ejection->pos[0 + tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos]+blockIdx.x*inst->nJobs])] + inst->resourcesAgent[ejection->pos[(ejection->sizeChain[term]-1) +tPos]*inst->mAgents + ((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])]>inst->capacity[((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs])])||(((int)sol->s[ejection->pos[0 + tPos] + blockIdx.x*inst->nJobs]) == ((int)sol->s[ejection->pos[(ejection->sizeChain[term]-1) + tPos]+blockIdx.x*inst->nJobs])))
				{
					aux=0;
				}
				for(i=0; i<inst->mAgents; i++)
				{
					if(res_aux[i]>inst->capacity[i])
					{
						aux=0;
						break;
					}
				}
				//printf("Ejection Chain, with delta equals the %d\n",ejection->delta[term]);
			}

		}
		while(aux==0);
		
		if(w==0){
			op_best = ejection->op[term];
			delta_best = ejection->delta[term];
			for (i=0;i<maxChain;i++){
				pos_best[i] =  ejection->pos[i + tPos];
			}
			size_best = ejection->sizeChain[term];

		}else{
			if(ejection->delta[term]<delta_best){
				op_best = ejection->op[term];
				delta_best = ejection->delta[term];
				for (i=0;i<maxChain;i++){
					pos_best[i] =  ejection->pos[i + tPos];
				}
				size_best = ejection->sizeChain[term];
			}

		}
	}
	ejection->op[term] = op_best;
	ejection->delta[term] = delta_best;
	for (i=0;i<maxChain;i++){
		ejection->pos[i + tPos] = pos_best[i];
	}
	ejection->sizeChain[term] = size_best;
}






Solution* createGPUsolution(Solution* h_solution,TnJobs nJobs, TmAgents mAgents)
{
	size_t size_solution = sizeof(Solution)
                        		   + sizeof(TcostFinal)*nBlocks
                        		   + sizeof(Ts)*(nJobs*nBlocks) //vector s
                        		   + sizeof(TresUsage)*(mAgents*nBlocks); // vector resUsage



	Solution *d_sol;
	gpuMalloc((void**)&d_sol, size_solution);
	gpuMemset(d_sol,0,size_solution);
	h_solution->costFinal = (TcostFinal*)(d_sol+1);
	h_solution->s = (Ts*)(h_solution->costFinal + nBlocks);
	h_solution->resUsage = (TresUsage*)(h_solution->s + (nJobs*nBlocks));
	gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);
	return d_sol;
}

EjectionChain* createGPUejection(EjectionChain* h_ejection,TnJobs nJobs, TmAgents mAgents)
{
	size_t size_ejection = sizeof(EjectionChain)
                        		   + sizeof(Tpos)*(nBlocks*nThreads*maxChain)
                        		   + sizeof(Top)*(nBlocks*nThreads)
                        		   + sizeof(TSizeChain)*(nBlocks*nThreads)
                        		   + sizeof(Tdelta)*(nBlocks*nThreads);
	EjectionChain *d_ejection;
	gpuMalloc((void**)&d_ejection, size_ejection);
	gpuMemset(d_ejection,0,size_ejection);
	h_ejection->pos=(Tpos*)(d_ejection + 1);
	h_ejection->op = (Top*)(h_ejection->pos+ (nBlocks*nThreads*maxChain));
	h_ejection->sizeChain = (TSizeChain*)(h_ejection->op + (nBlocks*nThreads));
	h_ejection->delta = (Tdelta*)(h_ejection->sizeChain + (nBlocks*nThreads));
	gpuMemcpy(d_ejection, h_ejection, size_ejection, hipMemcpyHostToDevice);
	return d_ejection;
}
